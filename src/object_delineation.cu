#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <iostream>
#include <vector>
#include <string>
#include <sstream>
#include <unordered_set>
#include <cmath>
#include <chrono>
#include <functional>

#ifdef USE_GTEST
#include <random>
#include <gtest/gtest.h>
#endif

#ifdef STANDALONE_RUN
#include <random>
#endif

class GeometryFactory {
public:
    GeometryFactory() {}
};

class BitArray {
public:
    BitArray(int size) : size_(size) {
        bits_.resize((size + 63) / 64, 0ULL);
    }
    inline bool get(int i) const {
        int idx = i / 64;
        int offset = i % 64;
        return (bits_[idx] & (1ULL << offset)) != 0;
    }
    inline void set(int i, bool value) {
        int idx = i / 64;
        int offset = i % 64;
        if (value) bits_[idx] |= (1ULL << offset);
        else bits_[idx] &= ~(1ULL << offset);
    }
    inline int countOnes() const {
        int c = 0;
        for (uint64_t b : bits_) c += __builtin_popcountll(b);
        return c;
    }
private:
    int size_;
    std::vector<uint64_t> bits_;
};

struct Vertex {
    int x;
    int y;
    Vertex* next;
    bool visited;
    Vertex(int xx, int yy, Vertex* nxt = nullptr) : x(xx), y(yy), next(nxt), visited(false) {}
};

class LiteList {
public:
    LiteList(const std::vector<short>& xs, const std::vector<short>& ys) : xs_(xs), ys_(ys) {}
    int numPoints() const { return xs_.size(); }
    std::string toString() const {
        std::ostringstream oss;
        for (size_t i = 0; i < xs_.size(); ++i) oss << "(" << xs_[i] << "," << ys_[i] << ") ";
        return oss.str();
    }
    double area() const { return 0.0; }
    const std::vector<short>& xs() const { return xs_; }
    const std::vector<short>& ys() const { return ys_; }
private:
    std::vector<short> xs_;
    std::vector<short> ys_;
};

struct Candidate {
    int x;
    int y;
    int pixelType;
    bool valid;
};

class ObjectDelineation {
public:
    using TransformFn = std::function<std::pair<int, int>(double, double)>;
    ObjectDelineation(int r, int b, TransformFn t = nullptr)
        : resolution_(r), buffer_(b), transform_(t),
          occPixels_((r + 2*b)*(r + 2*b)) {}
    int pointOffset(int x, int y) const {
        if (x < -buffer_ || x >= resolution_ + buffer_ || y < -buffer_ || y >= resolution_ + buffer_) return -1;
        return (y + buffer_)*(resolution_ + 2*buffer_) + (x + buffer_);
    }
    std::pair<int,int> transformCoord(double x, double y) const {
        if (transform_) return transform_(x, y);
        else return { (int)std::round(x), (int)std::round(y) };
    }
    BitArray& occupiedPixels() { return occPixels_; }
    void markRectangularBlock(int xs, int xe, int ys, int ye) {
        for (int y = ys; y < ye; y++) {
            for (int x = xs; x < xe; x++) {
                int off = pointOffset(x,y);
                if (off != -1) occPixels_.set(off,true);
            }
        }
    }
    std::vector<LiteList> aggregateOccupiedPixels();
private:
    int resolution_, buffer_;
    TransformFn transform_;
    BitArray occPixels_;
};

__global__ void detectCandidatesKernel(const bool* occArray, Candidate* cands, int r, int b, int w) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= w*w) return;
    int x = (idx % w) - b;
    int y = (idx / w) - b;
    int s = r + 2*b;
    int off = (y+b)*s + (x+b);
    int a0 = (x>-b && y>-b && (off-s-1)>=0 && occArray[off-s-1]) ? 1 : 0;
    int a1 = (x<r+b-1 && y>-b && occArray[off-s]) ? 2 : 0;
    int a2 = (x>-b && y<r+b-1 && occArray[off-1]) ? 4 : 0;
    int a3 = (x<r+b-1 && y<r+b-1 && occArray[off]) ? 8 : 0;
    int pt = a0|a1|a2|a3;
    bool v = true;
    switch(pt){ case 0: case 3: case 5: case 10: case 12: case 15: v=false; }
    cands[idx].x=x; cands[idx].y=y; cands[idx].pixelType=pt; cands[idx].valid=v;
}

static std::vector<LiteList> linkCandidates(const std::vector<Candidate>& cands, int w, int b, int r) {
    std::vector<Vertex*> topVertices(w, nullptr);
    Vertex* leftVertex=nullptr;
    std::vector<Vertex*> corners;
    for(int row=0; row<w; row++){
        for(int col=0; col<w; col++){
            int idx=row*w+col;
            if(!cands[idx].valid) continue;
            int x=cands[idx].x, y=cands[idx].y, pt=cands[idx].pixelType;
            switch(pt){
                case 1:{ Vertex* nv=new Vertex(x,y,leftVertex); if(topVertices[col]) topVertices[col]->next=nv; topVertices[col]=nullptr; leftVertex=nullptr; break; }
                case 2:{ Vertex* nv=new Vertex(x,y,topVertices[col]); leftVertex=nv; topVertices[col]=nullptr; break; }
                case 4:{ Vertex* nv=new Vertex(x,y,nullptr); if(leftVertex) leftVertex->next=nv; topVertices[col]=nv; leftVertex=nullptr; break; }
                case 6:{ Vertex* nv1=new Vertex(x,y,topVertices[col]); Vertex* nv2=new Vertex(x,y,nullptr); if(leftVertex) leftVertex->next=nv2; leftVertex=nv1; topVertices[col]=nv2; break; }
                case 7:{ Vertex* nv=new Vertex(x,y,nullptr); topVertices[col]=nv; leftVertex=nv; corners.push_back(nv); break; }
                case 8:{ Vertex* nv=new Vertex(x,y,nullptr); topVertices[col]=nv; leftVertex=nv; corners.push_back(nv); break; }
                case 9:{ Vertex* nv1=new Vertex(x,y,leftVertex); Vertex* nv2=new Vertex(x,y,nullptr); if(topVertices[col]) topVertices[col]->next=nv1; leftVertex=nv2; topVertices[col]=nv2; corners.push_back(nv2); break; }
                case 11:{ Vertex* nv=new Vertex(x,y,leftVertex); topVertices[col]=nv; leftVertex=nullptr; break; }
                case 13:{ Vertex* nv=new Vertex(x,y,nullptr); if(topVertices[col]) topVertices[col]->next=nv; leftVertex=nv; topVertices[col]=nullptr; break; }
                case 14:{ Vertex* nv=new Vertex(x,y,topVertices[col]); if(leftVertex) leftVertex->next=nv; leftVertex=nullptr; topVertices[col]=nullptr; break; }
                default: break;
            }
        }
    }
    std::vector<LiteList> rings;
    for (Vertex* c : corners) {
        if(!c->visited){
            std::vector<short> xs; std::vector<short> ys;
            Vertex* s=c; Vertex* cur=c;
            do{ xs.push_back((short)cur->x); ys.push_back((short)cur->y); cur->visited=true; cur=cur->next; } while(cur && cur!=s);
            xs.push_back((short)s->x); ys.push_back((short)s->y);
            rings.push_back(LiteList(xs, ys));
        }
    }
    std::unordered_set<Vertex*> freed;
    for(Vertex* c: corners){
        if(!c) continue;
        if(freed.find(c)!=freed.end()) continue;
        Vertex* tmp=c;
        do{ freed.insert(tmp); tmp=tmp->next; } while(tmp && tmp!=c);
    }
    for(auto*v:freed) delete v;
    return rings;
}

std::vector<LiteList> ObjectDelineation::aggregateOccupiedPixels() {
    int w=resolution_+2*buffer_; 
    int n=w*w; 
    int g=(resolution_+2*buffer_)*(resolution_+2*buffer_);
    bool* occ; 
    hipMallocManaged(&occ,g*sizeof(bool));
    for(int i=0;i<g;i++) occ[i]=occPixels_.get(i);
    Candidate* d_cand; 
    hipMallocManaged(&d_cand,n*sizeof(Candidate));
    int tpb=256; 
    int bl=(n+tpb-1)/tpb;
    detectCandidatesKernel<<<bl,tpb>>>(occ, d_cand, resolution_, buffer_, w);
    hipDeviceSynchronize();
    std::vector<Candidate> cands(n);
    for(int i=0;i<n;i++) cands[i]=d_cand[i];
    hipFree(d_cand);
    hipFree(occ);
    return linkCandidates(cands,w,buffer_,resolution_);
}

#ifdef STANDALONE_RUN
int main(){
    int resolution=1000; 
    int buffer=0; 
    ObjectDelineation od(resolution, buffer, nullptr);
    int total=(resolution+2*buffer)*(resolution+2*buffer);
    for(int i=0;i<total;i++){ od.occupiedPixels().set(i,(i%2==0)); }
    auto st=std::chrono::high_resolution_clock::now();
    auto rings=od.aggregateOccupiedPixels();
    auto en=std::chrono::high_resolution_clock::now();
    long long ms=std::chrono::duration_cast<std::chrono::milliseconds>(en-st).count();
    std::cout<<"CUDA sliding window + serial linking took "<<ms<<" ms\n";
    std::cout<<"Number of rings detected: "<<rings.size()<<"\n";
    return 0;
}
#endif

#ifdef USE_GTEST
#include <random>
#include <gtest/gtest.h>

TEST(ObjectDelineationTest, PerformanceStressTest){
    int r=1000,b=0;
    ObjectDelineation od(r,b,nullptr);
    std::mt19937 rng(0);
    std::uniform_real_distribution<double> dist(0.0,1.0);
    int t=(r+2*b)*(r+2*b);
    for(int i=0;i<t;i++){
        if(dist(rng)<0.5) od.occupiedPixels().set(i,true);
    }
    auto st=std::chrono::high_resolution_clock::now();
    auto rings=od.aggregateOccupiedPixels();
    auto en=std::chrono::high_resolution_clock::now();
    long long ms=std::chrono::duration_cast<std::chrono::milliseconds>(en-st).count();
    std::cout<<"Aggregation took "<<ms<<" ms\n";
    EXPECT_GT(rings.size(),0);
}

TEST(ObjectDelineationTest, PointOutside){
    int r=100,b=2;
    ObjectDelineation od(r,b,nullptr);
    int off=od.pointOffset(-10,-10);
    EXPECT_EQ(off,-1);
}

int main(int argc,char**argv){
    ::testing::InitGoogleTest(&argc,argv);
    return RUN_ALL_TESTS();
}
#endif
